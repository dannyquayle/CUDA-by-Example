#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"../common/book.h"
#include<stdio.h>

#define N (1024 * 1024)

__global__ void kernel(int* a, int* b, int* c) {
    int tid = blockIdx.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x;
    }
}

int main() {
    // 1. claim variables of cpu and gpu
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    // 2. cpu memory application of cpu variables
    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    // 3. initialize cpu varibales
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = 2 * i;
    }

    // 4. gpu memory application of gpu variables
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));

    // 5. memory copy for gpu variables from cpu to gpu
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    // 6. exec the kernel
    dim3 gridDim(128);
    dim3 blockDim(1);
    kernel<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c);

    // 7. memory copy for cpu variables from gpu to cpu
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    // 8. result check
    bool result = true;
    for (int i = 0; i < N; i++)
    {
        if (a[i] + b[i] != c[i]) {
            printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
            result = false;
            break;
        }
    }
    if (result) printf("WE DID IT!\n");

    // 9. free both the memory of cpu and gpu variables
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    free(a);
    free(b);
    free(c);

    return 0;
}